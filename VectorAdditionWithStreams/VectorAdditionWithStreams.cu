#include "hip/hip_runtime.h"
#include <wb.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
				        }                                                                     \
			    } while(0)

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < len) {
		out[i] = in1[i] + in2[i];
	}
}

int main(int argc, char **argv) {
	wbArg_t args;
	int inputLength;
	float *hostInput1;
	float *hostInput2;
	float *hostOutput;
	float *deviceInput1;
	float *deviceInput2;
	float *deviceOutput;

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput1 = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
	hostInput2 = (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
	hostOutput = (float *)malloc(inputLength * sizeof(float));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", inputLength);

	wbTime_start(GPU, "Allocating GPU memory.");
	wbCheck(hipMalloc((void **)&deviceInput1, inputLength*sizeof(float)));
	wbCheck(hipMalloc((void **)&deviceInput2, inputLength*sizeof(float)));
	wbCheck(hipMalloc((void **)&deviceOutput, inputLength*sizeof(float)));
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	wbCheck(hipMemcpy(deviceInput1, hostInput1, inputLength*sizeof(float), hipMemcpyHostToDevice));
	wbCheck(hipMemcpy(deviceInput2, hostInput2, inputLength*sizeof(float), hipMemcpyHostToDevice));
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	dim3 dimGrid((inputLength - 1) / 256 + 1, 1, 1);
	dim3 dimBlock(256, 1, 1);
	wbTime_start(Compute, "Performing CUDA computation");
	vecAdd << <dimGrid, dimBlock >> >(deviceInput1, deviceInput2, deviceOutput, inputLength);
	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	wbCheck(hipMemcpy(hostOutput, deviceOutput, inputLength*sizeof(float), hipMemcpyDeviceToHost));
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	wbCheck(hipFree(deviceInput1));
	wbCheck(hipFree(deviceInput2));
	wbCheck(hipFree(deviceOutput));
	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, hostOutput, inputLength);

	free(hostInput1);
	free(hostInput2);
	free(hostOutput);

	return 0;
}
