#include "hip/hip_runtime.h"
#include <wb.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define wbCheck(stmt)                                                          \
  do {                                                                         \
    hipError_t err = stmt;                                                    \
    if (err != hipSuccess) {                                                  \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                              \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));           \
      return -1;                                                               \
	    }                                                                          \
    } while (0)

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
		int numAColumns, int numBRows, int numBColumns,
		int numCRows, int numCColumns) {
	int cCol = blockDim.x*blockIdx.x + threadIdx.x;
	int cRow = blockDim.y*blockIdx.y + threadIdx.y;	

	if (cCol < numCColumns && cRow < numCRows) {		
		float cValue = 0;
		for (int i = 0; i < numAColumns; ++i) {
			cValue += A[cRow*numAColumns + i] * B[i*numBColumns + cCol];
		}
		C[cRow*numCColumns + cCol] = cValue;
	}
}

int main(int argc, char **argv) {
	wbArg_t args;
	float *hostA; // The A matrix
	float *hostB; // The B matrix
	float *hostC; // The output C matrix
	float *deviceA;
	float *deviceB;
	float *deviceC;
	int numARows;    // number of rows in the matrix A
	int numAColumns; // number of columns in the matrix A
	int numBRows;    // number of rows in the matrix B
	int numBColumns; // number of columns in the matrix B
	int numCRows;    // number of rows in the matrix C (you have to set this)
	int numCColumns; // number of columns in the matrix C (you have to set this)

	args = wbArg_read(argc, argv);

	wbTime_start(Generic, "Importing data and creating memory on host");
	hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
	hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
	//numAColumns must be == numBRows for valid multiplication. matrix product of [A(n x m)][B(m x p)] = [C(n x p)]
	numCRows = numARows;
	numCColumns = numBColumns;
	hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));
	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
	wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

	wbTime_start(GPU, "Allocating GPU memory.");
	wbCheck(hipMalloc((void **)&deviceA, numARows*numAColumns*sizeof(float)));
	wbCheck(hipMalloc((void **)&deviceB, numBRows*numBColumns*sizeof(float)));
	wbCheck(hipMalloc((void **)&deviceC, numCRows*numCColumns*sizeof(float)));
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	wbCheck(hipMemcpy(deviceA, hostA, numARows*numAColumns*sizeof(float), hipMemcpyHostToDevice));
	wbCheck(hipMemcpy(deviceB, hostB, numBRows*numBColumns*sizeof(float), hipMemcpyHostToDevice));
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	dim3 dimGrid((numCColumns - 1) / 16 + 1, (numCRows - 1) / 16 + 1, 1);
	dim3 dimBlock(16, 16, 1);
	wbTime_start(Compute, "Performing CUDA computation");
	matrixMultiply<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
	wbCheck(hipGetLastError());
	hipDeviceSynchronize();
	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	wbCheck(hipMemcpy(hostC, deviceC, numCRows*numCColumns*sizeof(float), hipMemcpyDeviceToHost));
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	wbCheck(hipFree(deviceA));
	wbCheck(hipFree(deviceB));
	wbCheck(hipFree(deviceC));
	wbTime_stop(GPU, "Freeing GPU Memory");

	wbSolution(args, hostC, numCRows, numCColumns);

	free(hostA);
	free(hostB);
	free(hostC);

	return 0;
}
